#include "hip/hip_runtime.h"
#include "max.cuh"

__global__ void MaxKernel(float *output, const float *in0, const float *in1, const int width, const int height) {
	const int i = blockDim.x * blockIdx.x + threadIdx.x;
	const int j = blockDim.y * blockIdx.y + threadIdx.y;
	if (i >= width || j >= height) {
		return;
	}

	float out_val = in0[(j * width) + i] > in1[(j * width) + i] ? in0[(j * width) + i] : in1[(j * width) + i];
	output[(j * width) + i] = out_val;
}

__global__ void MaxKernel3D(cnoise::Point* output, const cnoise::Point* in0, const cnoise::Point* in1, const int width, const int height) {
	const int i = blockDim.x * blockIdx.x + threadIdx.x;
	const int j = blockDim.y * blockIdx.y + threadIdx.y;
	if (i >= width || j >= height) {
		return;
	}
	float prev0, prev1;
	prev0 = in0[i + (j * width)].Value;
	prev1 = in0[i + (j * width)].Value;
	output[i + (j * width)].Value = (prev0 > prev1) ? prev0 : prev1;
}

void MaxLauncher(float *output, const float *in0, const float *in1, const int width, const int height) {

#ifdef CUDA_KERNEL_TIMING
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
#endif // CUDA_KERNEL_TIMING

	// Setup dimensions of kernel launch using occupancy calculator.
	int blockSize, minGridSize;
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, MaxKernel, 0, 0);
	dim3 block(blockSize, blockSize, 1);
	dim3 grid((width - 1) / blockSize + 1, (height - 1) / blockSize + 1, 1);
	MaxKernel<<<grid, block>>>(output, in0, in1, width, height);
	// Check for succesfull kernel launch
	cudaAssert(hipGetLastError());
	// Synchronize device
	cudaAssert(hipDeviceSynchronize());

#ifdef CUDA_KERNEL_TIMING
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float elapsed = 0.0f;
	hipEventElapsedTime(&elapsed, start, stop);
	printf("Kernel execution time in ms: %f\n", elapsed);
#endif // CUDA_KERNEL_TIMING

}

void MaxLauncher3D(cnoise::Point* output, const cnoise::Point* in0, const cnoise::Point* in1, const int width, const int height){

#ifdef CUDA_KERNEL_TIMING
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
#endif // CUDA_KERNEL_TIMING

	// Setup dimensions of kernel launch using occupancy calculator.
	dim3 block(16, 16, 1);
	dim3 grid(width / block.x, width / block.y, 1);
	MaxKernel3D<<<grid, block >>>(output, in0, in1, width, height);
	// Check for succesfull kernel launch
	cudaAssert(hipGetLastError());
	// Synchronize device
	cudaAssert(hipDeviceSynchronize());

#ifdef CUDA_KERNEL_TIMING
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float elapsed = 0.0f;
	hipEventElapsedTime(&elapsed, start, stop);
	printf("Kernel execution time in ms: %f\n", elapsed);
#endif // CUDA_KERNEL_TIMING

}
