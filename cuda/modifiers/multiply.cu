#include "hip/hip_runtime.h"
#include "power.cuh"


__global__ void multiplyKernel(hipSurfaceObject_t out, hipSurfaceObject_t in, const int width, const int height, float factor) {
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i >= width || j >= height) {
		return;
	}

	float prev;
	surf2Dread(&prev, input, i * sizeof(float), j);

	float final_value;
	final_value = prev * factor;



	surf2Dwrite(final_value, out, i * sizeof(float), j);

}

void multiplyLauncher(hipSurfaceObject_t out, hipSurfaceObject_t in, const int width, const int height, float factor) {
#ifdef CUDA_TIMING_TESTS
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
#endif // CUDA_TIMING_TESTS

	// Setup dimensions of kernel launch. 
	dim3 threadsPerBlock(32, 32);
	dim3 numBlocks(width / threadsPerBlock.x, height / threadsPerBlock.y);


	multiplyKernel << <block, grid >> >(out, in, width, height, factor);


	// Check for succesfull kernel launch
	cudaAssert(hipGetLastError());
	// Synchronize device
	cudaAssert(hipDeviceSynchronize());
#ifdef CUDA_TIMING_TESTS
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float elapsed = 0.0f;
	hipEventElapsedTime(&elapsed, start, stop);
	printf("Kernel execution time in ms: %f\n", elapsed);
#endif // CUDA_TIMING_TESTS
	// If this completes, kernel is done and "output" contains correct data.
}