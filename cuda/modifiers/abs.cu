#include "hip/hip_runtime.h"
#include "abs.cuh"


__global__ void absKernel(hipSurfaceObject_t out, hipSurfaceObject_t in, const int width, const int height) {
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i >= width || j >= height) {
		return;
	}

	float prev;
	surf2Dread(&prev, input, i * sizeof(float), j);

	float final_value;

	if (prev <= 0)
	{
		final_value = -prev;
	}

	else
	{
		final_value = prev;
	}
	

	surf2Dwrite(final_value, out, i * sizeof(float), j);

}

void absLauncher(hipSurfaceObject_t out, hipSurfaceObject_t in, const int width, const int height) {
#ifdef CUDA_TIMING_TESTS
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
#endif // CUDA_TIMING_TESTS

	// Setup dimensions of kernel launch. 

	// Use occupancy calc to find optimal sizes.
	int blockSize, minGridSize;
#ifdef CUDA_TIMING_TESTS
	hipEventRecord(start);
#endif // CUDA_TIMING_TESTS
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, (void*)absKernel, 0, 0); //???
	dim3 block(blockSize, blockSize, 1);
	dim3 grid((width - 1) / blockSize + 1, (height - 1) / blockSize + 1, 1);
	absKernel << <block, grid >> >(out, in, width, height);
	// Check for succesfull kernel launch
	cudaAssert(hipGetLastError());
	// Synchronize device
	cudaAssert(hipDeviceSynchronize());
#ifdef CUDA_TIMING_TESTS
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float elapsed = 0.0f;
	hipEventElapsedTime(&elapsed, start, stop);
	printf("Kernel execution time in ms: %f\n", elapsed);
#endif // CUDA_TIMING_TESTS
	// If this completes, kernel is done and "output" contains correct data.
}