#include "hip/hip_runtime.h"
#include "perlin_tex.cuh"
#include "../vector_operators.cuh"

__device__ float perlin2d_tex(hipTextureObject_t permutation, hipTextureObject_t gradient, const float px, const float py, const int seed) {
	// Calculate 2D integer coordinates and fractional component 
	float2 i = make_float2(floorf(px), floorf(py));
	float2 f = make_float2(px - i.x, py - i.y);

	// Get weights.
	float2 w;
	w.x = f.x * f.x * f.x * (f.x * (f.x * 6.0f - 15.0f) + 10.0f);
	w.y = f.y * f.y * f.y * (f.y * (f.y * 6.0f - 15.0f) + 10.0f);
	float4 w4 = make_float4(1.0f, w.x, w.y, w.x * w.y);

	// Get four randomly permutated indices from the noise lattice nearest "point"
	// and offset them by the seed.
	uchar4 tmp = tex2D<uchar4>(permutation, i.x + 0.50f, i.y + 0.50f);
	float4 perm = make_float4(tmp.x, tmp.y, tmp.z, tmp.w);
	perm = perm + seed;

	// Permute the fourst indices again and get the 2D gradient for each of
	// the four new coord-seed pairs.
	float4 gLeft, gRight;
	uchar4 tmp0 = tex2D<uchar4>(gradient, perm.x + 0.50f, perm.y + 0.50f);
	gLeft = make_float4(tmp0.x, tmp0.y, tmp0.z, tmp0.w);
	gLeft = gLeft * 2.0f;
	gLeft = gLeft - 1.0f;
	uchar4 tmp1 = tex2D<uchar4>(gradient, perm.z + 0.50f, perm.w + 0.50f);
	gRight = make_float4(tmp1.x, tmp1.y, tmp1.z, tmp1.w);
	gRight = gRight * 2.0f;
	gRight = gRight - 1.0f;

	// Evaluate gradients at four lattice points.
	float nLeftTop = dot(make_float2(gLeft.x, gLeft.y), f);
	float nRightTop = dot(make_float2(gRight.x, gRight.y), f + make_float2(-1.0f, 0.0f));
	float nLeftBottom = dot(make_float2(gLeft.z, gLeft.w), f + make_float2(0.0f, -1.0f));
	float nRightBottom = dot(make_float2(gRight.z, gRight.w), f + make_float2(-1.0f, -1.0f));

	// Blend gradients.
	float4 gradientBlend = make_float4(nLeftTop, nRightTop - nLeftTop, nLeftBottom - nLeftTop,
		nLeftTop - nRightTop - nLeftBottom + nRightBottom);
	float n = dot(gradientBlend, w4);

	// Return value.
	return (n * 1.5f) / (2.5f);
	//return n * 1.530734f;
}

__device__ float FBM2d_tex(hipTextureObject_t permutation, hipTextureObject_t gradient, float px, float py, const float freq, const float lacun, const float persist, const int init_seed, const int octaves) {
	float amplitude = 1.0f;
	// Scale point by freq
	px *= freq;
	py *= freq;
	// TODO: Seeding the function is currently pointless and doesn't actually do anything.
	// Use loop for octav-ing
	float result = 0.0f;
	for (size_t i = 0; i < octaves; ++i) {
		int seed = (init_seed + i) & 0xffffffff;
		result += perlin2d_tex(permutation, gradient, px, py, seed) * amplitude;
		// Modify vars for next octave.
		px *= lacun;
		py *= lacun;
		amplitude *= persist;
	}

	return result;
}

__global__ void texFBMKernel(hipSurfaceObject_t output, hipTextureObject_t permutation, hipTextureObject_t gradient, const int width, const int height, const float2 origin, const float freq, const float lacun, const float persist, const int seed, const int octaves) {
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i < width && j < height) {
		float2 p = make_float2(origin.x + i, origin.y + j);
		// Call billow function
		float val = FBM2d_tex(permutation, gradient, p.x, p.y, freq, lacun, persist, seed, octaves);
		// Write val to the surface
		surf2Dwrite(val, output, i * sizeof(float), j);
	}
}

void texFBMLauncher(hipSurfaceObject_t output, hipTextureObject_t permutation, hipTextureObject_t gradient, const int width, const int height, const float2 origin, const float freq, const float lacun, const float persist, const int seed, const int octaves){
#ifdef CUDA_TIMING_TESTS
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
#endif // CUDA_TIMING_TESTS

	dim3 threadsPerBlock(8, 8);
	dim3 numBlocks(width / threadsPerBlock.x, height / threadsPerBlock.y);
	texFBMKernel<<<numBlocks, threadsPerBlock>>>(output, permutation, gradient, width, height, origin, freq, lacun, persist, seed, octaves);
	// Check for succesfull kernel launch
	cudaAssert(hipGetLastError());
	// Synchronize device
	cudaAssert(hipDeviceSynchronize());

#ifdef CUDA_TIMING_TESTS
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float elapsed = 0.0f;
	hipEventElapsedTime(&elapsed, start, stop);
	printf("Kernel execution time in ms: %f\n", elapsed);
#endif // CUDA_TIMING_TESTS
}