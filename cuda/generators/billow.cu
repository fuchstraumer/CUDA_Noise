#include "hip/hip_runtime.h"
#include "billow.cuh"
#include "..\..\cpp\modules\generators\Billow.h"


__device__ float billow2D(float2 point, float freq, float lacun, float persist, int init_seed, int octaves) {
	// Will be incremented upon.
	float result = 0.0f;
	float amplitude = 1.0f;
	float val;
	// Scale point by freq
	point.x = point.x * freq;
	point.y = point.y * freq;
	// TODO: Seeding the function is currently pointless and doesn't actually do anything.
	// Use loop for octav-ing
	for (size_t i = 0; i < octaves; ++i) {
		int seed = (init_seed + i) & 0xffffffff;
		val = perlin2d(point, freq, seed);
		val = fabsf(val);
		result += val * amplitude;
		// Modify vars for next octave.
		freq *= lacun;
		point.x *= freq;
		point.y *= freq;
		amplitude *= persist;
	}
	// float tmp = result / 100.0f;
	// * // 
	return result;
}



__global__ void Billow2DKernel(hipSurfaceObject_t out, int width, int height, float2 origin, float freq, float lacun, float persist, int seed, int octaves) {
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i >= width || j >= height) {
		return;
	}

	float x, y;
	x = i + origin.x;
	y = j + origin.y;
	float2 p = make_float2(x, y);
	// Call billow function
	float val = billow2D(p, freq, lacun, persist, seed, octaves);

	// Write val to the surface
	surf2Dwrite(val, out, i * sizeof(float), j);
}




void BillowLauncher(hipSurfaceObject_t out, int width, int height, float2 origin, float freq, float lacun, float persist, int seed, int octaves) {
#ifdef CUDA_TIMING_TESTS
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
#endif // CUDA_TIMING_TESTS

	// Setup dimensions of kernel launch. 
#ifdef CUDA_TIMING_TESTS
	hipEventRecord(start);
#endif // CUDA_TIMING_TESTS
	dim3 threadsPerBlock(32, 32);
	dim3 numBlocks(width / threadsPerBlock.x, height / threadsPerBlock.y);
	Billow2DKernel<<<numBlocks,threadsPerBlock>>>(out, width, height, origin, freq, lacun, persist, seed, octaves);
	// Check for succesfull kernel launch
	cudaAssert(hipGetLastError());
	// Synchronize device
	cudaAssert(hipDeviceSynchronize());
#ifdef CUDA_TIMING_TESTS
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float elapsed = 0.0f;
	hipEventElapsedTime(&elapsed, start, stop);
	printf("Kernel execution time in ms: %f\n", elapsed);
#endif // CUDA_TIMING_TESTS
	// If this completes, kernel is done and "output" contains correct data.
}


/*

	
	Following are BROKEN Simplex methods. Output is HEAVILY artifacted. There is no implementation of Simplex
	module construction in C++ anymore, either, as there were far too many errors here.


*/

__device__ float billow2D_S(float2 point, float freq, float lacun, float persist, int init_seed, int octaves) {
	float result = 0.0f;
	float amplitude = 1.0f;
	float val;
	// Scale starting point by frequency.
	point.x = point.x * freq;
	point.y = point.y * freq;
	// Use loop for fractal octave bit
	for (size_t i = 0; i < octaves; ++i) {
		val = simplex2d(point, freq);
		val = fabsf(val);
		result += val * amplitude;
		freq *= lacun;
		point.x *= freq;
		point.y *= freq;
		amplitude *= persist;
	}
	//result /= 100.0f;
	return result;
}

__global__ void Billow2DKernelSimplex(hipSurfaceObject_t out, int width, int height, float2 origin, float freq, float lacun, float persist, int seed, int octaves){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i >= width || j >= height) {
		return;
	}

	float x, y;
	x = i + origin.x;
	y = j + origin.y;
	float2 p = make_float2(x, y);
	// Call billow function
	float val = billow2D_S(p, freq, lacun, persist, seed, octaves);

	// Write val to the surface
	surf2Dwrite(val, out, i * sizeof(float), j);
}


void BillowSimplexLauncher(hipSurfaceObject_t out, int width, int height, float2 origin, float freq, float lacun, float persist, int seed, int octaves){
	size_t heap, stack;
	hipDeviceGetLimit(&heap, hipLimitMallocHeapSize);
	hipDeviceGetLimit(&stack, hipLimitStackSize);
	hipDeviceSetLimit(hipLimitMallocHeapSize, heap);
	hipDeviceSetLimit(hipLimitStackSize, stack);
#ifdef CUDA_TIMING_TESTS
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
#endif // CUDA_TIMING_TESTS
	// Setup dimensions of kernel launch. 
#ifdef CUDA_TIMING_TESTS
	hipEventRecord(start);
#endif // CUDA_TIMING_TESTS
	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks(width / threadsPerBlock.x, height / threadsPerBlock.y);
	Billow2DKernelSimplex<<<numBlocks, threadsPerBlock>>>(out, width, height, origin, freq, lacun, persist, seed, octaves);
	// Check for succesfull kernel launch
	cudaAssert(hipGetLastError());
	// Synchronize device
	cudaAssert(hipDeviceSynchronize());
#ifdef CUDA_TIMING_TESTS
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float elapsed = 0.0f;
	hipEventElapsedTime(&elapsed, start, stop);
	printf("Kernel execution time in ms: %f\n", elapsed);
#endif // CUDA_TIMING_TESTS
	// If this completes, kernel is done and "output" contains correct data.
}

