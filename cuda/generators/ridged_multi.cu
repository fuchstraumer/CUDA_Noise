#include "hip/hip_runtime.h"
#include "ridged_multi.cuh"

__device__ float Ridged2D_Simplex(float2 point, const float freq, const float lacun, const float persist, const int init_seed, const int octaves) {
	float result = 0.0f;
	float amplitude = 1.0f;
	// Scale starting point by frequency.
	point.x = point.x * freq;
	point.y = point.y * freq;
	// Use loop for fractal octave bit
	for (size_t i = 0; i < octaves; ++i) {
		int seed = (init_seed + i) & 0xffffffff;
		result += (1.0f - fabsf(simplex2d(point.x, point.y, seed, nullptr))) * amplitude;
		point.x *= lacun;
		point.y *= lacun;
		amplitude *= persist;
	}
	return result;
}

__device__ float Ridged2D(float2 point, const float freq, const float lacun, const float persist, const int init_seed, const int octaves) {
	// Will be incremented upon.
	float result = 0.0f;
	float amplitude = 1.0f;
	// Scale point by freq
	point.x = point.x * freq;
	point.y = point.y * freq;
	// TODO: Seeding the function is currently pointless and doesn't actually do anything.
	// Use loop for octav-ing
	for (size_t i = 0; i < octaves; ++i) {
		int seed = (init_seed + i) & 0xffffffff;
		result += (1.0f - fabsf(perlin2d(point.x, point.y, seed, nullptr)))* amplitude;
		// Modify vars for next octave.
		point.x *= lacun;
		point.y *= lacun;
		amplitude *= persist;
	}
	return result;
}

__device__ float Ridged3D(float3 p, const float freq, const float lacun, const float persist, const int init_seed, const int octaves) {
	float amplitude = 1.0f;
	float result = 0.0f;
	p *= freq;
	for (int i = 0; i < octaves; ++i) {
		int seed = (init_seed + i) & 0xffffffff;
		result += (1.0f - fabsf(simplex3d(p.x, p.y, p.z, seed, nullptr))) * amplitude;
		p *= lacun;
		amplitude *= persist;
	}
	return result;
}

__global__ void Ridged2DKernel(float* out, int width, int height, cnoise::noise_t noise_type, float2 origin, float freq, float lacun, float persist, int seed, int octaves) {
	const int i = blockDim.x * blockIdx.x + threadIdx.x;
	const int j = blockDim.y * blockIdx.y + threadIdx.y;
	if (i < width && j < height) {
		// Get offset pos.
		float2 p = make_float2(i + origin.x, j + origin.y);
		// Call ridged function
		float val;
		switch (noise_type) {
			case(cnoise::noise_t::PERLIN): {
				val = Ridged2D(p, freq, lacun, persist, seed, octaves);
				break;
			}
			case(cnoise::noise_t::SIMPLEX): {
				val = Ridged2D_Simplex(p, freq, lacun, persist, seed, octaves);
				break;
			}
		}
		// Write val to the surface
		out[(j * width) + i] = val;
	}
	
}

__global__ void Ridged3DKernel(cnoise::Point* coords, const int width, const int height, const float freq, const float lacun, const float persist, const int seed, const int octaves) {
	const int i = blockDim.x * blockIdx.x + threadIdx.x;
	const int j = blockDim.y * blockIdx.y + threadIdx.y;
	if (i >= width || j >= height) {
		return;
	}
	coords[i + (j * width)].Value = Ridged3D(coords[i + (j * width)].Position, freq, lacun, persist, seed, octaves);
}

void RidgedMultiLauncher(float* out, int width, int height, cnoise::noise_t noise_type, float2 origin, float freq, float lacun, float persist, int seed, int octaves) {

#ifdef CUDA_KERNEL_TIMING
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
#endif // CUDA_KERNEL_TIMING

	hipFuncAttributes attr;
	hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(Ridged2DKernel));
	dim3 threadsPerBlock(8, 8);
	dim3 numBlocks(width / threadsPerBlock.x, height / threadsPerBlock.y);
	Ridged2DKernel<<<numBlocks, threadsPerBlock>>>(out, width, height, noise_type, origin, freq, lacun, persist, seed, octaves);
	hipError_t err = hipGetLastError();
	cudaAssert(err);
	// Synchronize device
	err = hipDeviceSynchronize();
	cudaAssert(err);

#ifdef CUDA_KERNEL_TIMING
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float elapsed = 0.0f;
	hipEventElapsedTime(&elapsed, start, stop);
	printf("Ridged-multi Kernel execution time in ms: %f\n", elapsed);
#endif // CUDA_KERNEL_TIMING

	// If this completes, kernel is done and "output" contains correct data.
}

void RidgedMultiLauncher3D(cnoise::Point* coords, const int width, const int height, const float freq, const float lacun, const float persist, const int seed, const int octaves){

#ifdef CUDA_KERNEL_TIMING
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
#endif // CUDA_KERNEL_TIMING

	hipFuncAttributes attr;
	hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(Ridged2DKernel));
	dim3 threadsPerBlock(8, 8);
	dim3 numBlocks(width / threadsPerBlock.x, height / threadsPerBlock.y);
	Ridged3DKernel<<<numBlocks, threadsPerBlock >>>(coords, width, height, freq, lacun, persist, seed, octaves);
	hipError_t err = hipGetLastError();
	cudaAssert(err);
	// Synchronize device
	err = hipDeviceSynchronize();
	cudaAssert(err);

#ifdef CUDA_KERNEL_TIMING
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float elapsed = 0.0f;
	hipEventElapsedTime(&elapsed, start, stop);
	printf("Kernel execution time in ms: %f\n", elapsed);
#endif // CUDA_KERNEL_TIMING

}
