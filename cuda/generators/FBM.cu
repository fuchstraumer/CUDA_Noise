#include "hip/hip_runtime.h"
#include "FBM.cuh"
#include "..\..\cpp\modules\generators\FBM.h"

__device__ float FBM2d_Simplex(float2 point, float freq, float lacun, float persist, int init_seed, float octaves) {
	// Will be incremented upon.
	float result = 0.0f;
	float amplitude = 1.0f;
	float val;
	// Scale point by freq
	point.x = point.x * freq;
	point.y = point.y * freq;
	// TODO: Seeding the function is currently pointless and doesn't actually do anything.
	// Use loop for octav-ing
	for (size_t i = 0; i < octaves; ++i) {
		int seed = (init_seed + i) & 0xffffffff;
		val = simplex2d(point, freq);
		result += val * amplitude;
		// Modify vars for next octave.
		freq *= lacun;
		point.x *= freq;
		point.y *= freq;
		amplitude *= persist;
	}
	// float tmp = result / 100.0f;
	// * // 
	return result;
}

__device__ float FBM2d(float2 point, float freq, float lacun, float persist, int init_seed, float octaves) {
	// Will be incremented upon.
	float result = 0.0f;
	float amplitude = 1.0f;
	float val;
	// Scale point by freq
	point.x = point.x * freq;
	point.y = point.y * freq;
	// TODO: Seeding the function is currently pointless and doesn't actually do anything.
	// Use loop for octav-ing
	for (size_t i = 0; i < octaves; ++i) {
		int seed = (init_seed + i) & 0xffffffff;
		val = perlin2d(point, freq, seed);
		result += val * amplitude;
		// Modify vars for next octave.
		freq *= lacun;
		point.x *= freq;
		point.y *= freq;
		amplitude *= persist;
	}
	// float tmp = result / 100.0f;
	// * // 
	return result;
}

__global__ void FBM2DKernel(hipSurfaceObject_t out, int width, int height, noise_t noise_type, float2 origin, float freq, float lacun, float persist, int seed, int octaves){
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i >= width || j >= height) {
		return;
	}

	float x, y;
	x = i + origin.x;
	y = j + origin.y;
	float2 p = make_float2(x, y);
	// Call billow function
	float val;
	switch (noise_type) {
		case(noise_t::PERLIN): {
			val = FBM2d(p, freq, lacun, persist, seed, octaves);
		}
		case(noise_t::SIMPLEX): {
			val = FBM2d_Simplex(p, freq, lacun, persist, seed, octaves);
		}
	}

	// Write val to the surface
	surf2Dwrite(val, out, i * sizeof(float), j);
}

void FBM_Launcher(hipSurfaceObject_t out, int width, int height, noise_t noise_type, float2 origin, float freq, float lacun, float persist, int seed, int octaves){
#ifdef CUDA_TIMING_TESTS
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
#endif // CUDA_TIMING_TESTS

	dim3 threadsPerBlock(32, 32);
	dim3 numBlocks(width / threadsPerBlock.x, height / threadsPerBlock.y);
	FBM2DKernel<<<numBlocks, threadsPerBlock>>>(out, width, height, noise_type, origin, freq, lacun, persist, seed, octaves);
	// Check for succesfull kernel launch
	cudaAssert(hipGetLastError());
	cudaAssert(hipDeviceSynchronize());
	// Synchronize device
	cudaAssert(hipDeviceSynchronize());

#ifdef CUDA_TIMING_TESTS
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float elapsed = 0.0f;
	hipEventElapsedTime(&elapsed, start, stop);
	printf("Kernel execution time in ms: %f\n", elapsed);
#endif // CUDA_TIMING_TESTS

	// If this completes, kernel is done and "output" contains correct data.
}




