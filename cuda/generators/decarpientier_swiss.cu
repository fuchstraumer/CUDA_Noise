#include "hip/hip_runtime.h"
#include "decarpientier_swiss.cuh"

__device__ float d_swiss_simplex(float2 point, const float freq, const float lacun, const float persist, const int octaves) {
	// Will be incremented upon.
	float result = 0.0f;
	float amplitude = 1.0f;
	// Scale point by freq
	point.x = point.x * freq;
	point.y = point.y * freq;
	// TODO: Seeding the function is currently pointless and doesn't actually do anything.
	// Use loop for octav-ing
	float warp = 0.1f;
	float dx_sum = 0.0f, dy_sum = 0.0f;
	for (size_t i = 0; i < octaves; ++i) {
		float2 dx_dy; 
		volatile float n = simplex2d(point, &dx_dy);
		result += (1.0f - fabsf(n)) * amplitude;
		dx_sum += amplitude * dx_dy.x * -n;
		dy_sum += amplitude * dx_dy.y * -n;
		// Modify vars for next octave.
		point.x *= lacun;
		point.y *= lacun;
		point.x += (warp * dx_sum);
		point.y += (warp * dy_sum);
		amplitude *= persist * __saturatef(result);
	}

	return result;
}

__device__ float d_swiss_perlin(float px, float py, const float freq, const float lacun, const float persist, const int init_seed, const int octaves) {
	float amplitude = 1.0f;
	// Scale point by freq
	float2 point = make_float2(px * freq, py * freq);
	float warp = 0.1f;
	// TODO: Seeding the function is currently pointless and doesn't actually do anything.
	// Use loop for octav-ing
	float result = 0.0f;
	float dx_sum = 0.0f, dy_sum = 0.0f;
	for (size_t i = 0; i < octaves; ++i) {
		int seed = (init_seed + i) & 0xffffffff;
		float n = perlin2d(point, seed);
		result += (1.0f - fabsf(n)) * amplitude;
		dx_sum += amplitude * perlin2d_dx(point, seed) * -n;
		dy_sum += amplitude * perlin2d_dy(point, seed) * -n;
		// Modify vars for next octave.
		point.x += (warp * dx_sum);
		point.y += (warp * dy_sum);
		point.x *= lacun;
		point.y *= lacun;
		amplitude *= persist * __saturatef(result);
	}
	return result;
}

__global__ void d_swiss_kernel(hipSurfaceObject_t out, int width, int height, noise_t noise_type, float2 origin, float freq, float lacun, float persist, int seed, int octaves) {
	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < width && j < height) {
		// Call noise function
		float val;
		switch (noise_type) {
		case noise_t::PERLIN:
			val = d_swiss_perlin(origin.x + i, origin.y + j, freq, lacun, persist, seed, octaves);
			break;
		case noise_t::SIMPLEX:
			val = d_swiss_simplex(make_float2(origin.x + i, origin.y + j), freq, lacun, persist, octaves);
			break;
		}
		// Write val to the surface
		surf2Dwrite(val, out, i * sizeof(float), j);
	}
}

void DecarpientierSwissLauncher(hipSurfaceObject_t out, int width, int height, noise_t noise_type, float2 origin, float freq, float lacun, float persist, int seed, int octaves) {
#ifdef CUDA_TIMING_TESTS
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
#endif // CUDA_TIMING_TESTS

	hipFuncAttributes attr;
	hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(d_swiss_kernel));
	dim3 threadsPerBlock(8, 8);
	dim3 numBlocks(width / threadsPerBlock.x, height / threadsPerBlock.y);
	d_swiss_kernel<<<numBlocks, threadsPerBlock>>>(out, width, height, noise_type, origin, freq, lacun, persist, seed, octaves);
	// Check for succesfull kernel launch
	cudaAssert(hipGetLastError());
	// Synchronize device
	cudaAssert(hipDeviceSynchronize());

#ifdef CUDA_TIMING_TESTS
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float elapsed = 0.0f;
	hipEventElapsedTime(&elapsed, start, stop);
	printf("Kernel execution time in ms: %f\n", elapsed);
#endif // CUDA_TIMING_TESTS

	// If this completes, kernel is done and "output" contains correct data.
}